#include "hip/hip_runtime.h"
/*
HOW TO USE?
nvcc -I "cuda-samples\Common" -o output_file.exe mysampleforcudacopy.cu
Of course you can use output_file to replace output_file.exe in linux
Author : Eloim, ewppple1999@qq.com
*/

#include <cstdio>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

#define NUM_ELEMENTS 400000000

__global__ void add(int *a, int *b, int *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < NUM_ELEMENTS) {
        c[idx] = a[idx] + b[idx];
    }
}


int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("  I have %d GPUs.\n\n", deviceCount);

    if (deviceCount < 3) return 0;

    int *d_a;
    hipSetDevice(0);
    hipMalloc(&d_a, NUM_ELEMENTS * sizeof(int));
    hipMemset(d_a, 1, NUM_ELEMENTS * sizeof(int));
    hipStream_t stream1; hipEvent_t start1; hipEvent_t stop1;
    hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
    hipEventCreate(&start1); hipEventCreate(&stop1);
    // temp output the d_a
    /*
    int *host_buffer1 = new int[NUM_ELEMENTS];
    hipMemcpy(host_buffer1, d_a, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost);
    for (int j = 0; j < NUM_ELEMENTS; j++)
    {
        printf("%d ",host_buffer1[j]);
        if ( j % 10 == 9){
            printf("\n");
        }
        if (j % 400 == 399) break;
    }
    */

    int *d_b;
    hipSetDevice(1);
    hipMalloc(&d_b, NUM_ELEMENTS * sizeof(int));
    hipMemset(d_b, 2, NUM_ELEMENTS * sizeof(int));
    hipStream_t stream2; hipEvent_t start2; hipEvent_t stop2;
    hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
    hipEventCreate(&start2); hipEventCreate(&stop2);

    int *d_c;
    hipSetDevice(2);
    hipMalloc(&d_c, NUM_ELEMENTS * sizeof(int));
    hipStream_t stream3; hipEvent_t start3; hipEvent_t stop3;
    hipStreamCreateWithFlags(&stream3, hipStreamNonBlocking);
    hipEventCreate(&start3); hipEventCreate(&stop3);

    int *tmp_a, *tmp_b;
    hipMalloc(&tmp_a, NUM_ELEMENTS * sizeof(int));
    hipMalloc(&tmp_b, NUM_ELEMENTS * sizeof(int));


    // copy
    hipEventRecord(start1, stream1);
    hipMemcpyAsync(tmp_a, d_a, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToDevice, stream1);
    hipEventRecord(stop1, stream1);
    hipStreamSynchronize(stream1);
    hipEventRecord(start2, stream2);
    hipMemcpyAsync(tmp_b, d_b, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToDevice, stream2);
    hipEventRecord(stop2, stream2);
    hipStreamSynchronize(stream2);

    // add
    int threadsPerBlock = 256;
    int blocksPerGrid = (NUM_ELEMENTS + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid, threadsPerBlock>>>(tmp_a, tmp_b, d_c);

    // copy back
    hipEventRecord(start3, stream3);
    hipMemcpyAsync(d_a, d_c, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToDevice, stream3);
    hipEventRecord(stop3, stream3);
    hipStreamSynchronize(stream3);

    float time_ms1; float time_ms2; float time_ms3;
    hipEventElapsedTime(&time_ms1, start1, stop1);
    hipEventElapsedTime(&time_ms2, start2, stop2);
    hipEventElapsedTime(&time_ms3, start3, stop3);
    double time_s1 = time_ms1 / 1e3;
    double time_s2 = time_ms2 / 1e3;
    double time_s3 = time_ms3 / 1e3;

    double gb1 = NUM_ELEMENTS * sizeof(int) / (double)1e9;
    double gb2 = NUM_ELEMENTS * sizeof(int) / (double)1e9;
    double gb3 = NUM_ELEMENTS * sizeof(int) / (double)1e9;
    double bandwidth1 = gb1 / time_s1;
    double bandwidth2 = gb2 / time_s2;
    double bandwidth3 = gb3 / time_s3;

    printf("  OP1 use %6.02f second and datasize is %6.02f GB, bandwidth is %6.02f GB/s\n",time_s1, gb1, bandwidth1);
    printf("  OP2 use %6.02f second and datasize is %6.02f GB, bandwidth is %6.02f GB/s\n",time_s2, gb2, bandwidth2);
    printf("  OP3 use %6.02f second and datasize is %6.02f GB, bandwidth is %6.02f GB/s\n\n",time_s3, gb3, bandwidth3);

    int *h_result = new int[NUM_ELEMENTS];
    hipMemcpy(h_result, d_a, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost);


    for (int i = 0; i < NUM_ELEMENTS; i++)
    {
        printf("%d ",h_result[i]);
        if ( i % 10 == 9){
            printf("\n");
        }
        if (i % 400 == 399) break;
    }

    delete[] h_result;
    hipFree(d_a); hipFree(d_b); hipFree(d_c); hipFree(tmp_a); hipFree(tmp_b);
    hipEventDestroy(start1);hipEventDestroy(start2);hipEventDestroy(start3);
    hipEventDestroy(stop1);hipEventDestroy(stop2);hipEventDestroy(stop3);
    hipStreamDestroy(stream1);hipStreamDestroy(stream2);hipStreamDestroy(stream3);

    return 0;
}